#include "hip/hip_runtime.h"
#include "config.h"
#include "interaction_bound.h"
#include "number_of_bodies.h"
#include "planets.h"
#include "nbody_exception.h"

#include "thrust\device_vector.h"
#include "thrust\host_vector.h"
#include "thrust\generate.h"
#include "thrust\copy.h"

#include "hip/hip_runtime.h"
#include ""

#define THREADS_PER_BLOCK	256

// General 3D vector routines
__device__
vec_t	cross_product(const vec_t* v, const vec_t* u)
{
	vec_t result;

	result.x = v->y*u->z - v->z*u->y;
    result.y = v->z*u->x - v->x*u->z;
    result.z = v->x*u->y - v->y*u->x;

	return result;
}

inline __device__
var_t	dot_product(const vec_t* v, const vec_t* u)
{
	return v->x * u->x + v->y * u->y + v->z * u->z;
}

inline __device__
var_t	norm2(const vec_t* v)
{
	return SQR(v->x) + SQR(v->y) + SQR(v->z);
}

inline __device__
var_t	norm(const vec_t* v)
{
	return sqrt(norm2(v));
}

// Calculate acceleration caused by particle j on parrticle i 
__device__ 
vec_t calculate_grav_accel_pair(const vec_t ci, const vec_t cj, var_t mass, vec_t a)
{
	vec_t d;
	
	d.x = cj.x - ci.x;
	d.y = cj.y - ci.y;
	d.z = cj.z - ci.z;

	d.w = SQR(d.x) + SQR(d.y) + SQR(d.z);
	d.w = d.w * d.w * d.w;
	d.w =-K2 * mass / sqrt(d.w);

	a.x += d.x * d.w;
	a.y += d.y * d.w;
	a.z += d.z * d.w;

	return a;
}

__device__ 
vec_t circular_velocity(var_t mu, var_t r, var_t alpha)
{
	vec_t	result;

	var_t v		= sqrt(mu/r);
	result.x	=-v*sin(alpha);
	result.y	= v*cos(alpha);
	result.z	= 0.0;

	return result;
}

__device__
vec_t gas_velocity(var2_t eta, var_t mu, var_t r, var_t alpha)
{
	vec_t result = circular_velocity(mu, r, alpha);

	var_t v		 = sqrt(1.0 - 2.0*eta.x * pow(r, eta.y));
	result.x	*= v;
	result.y	*= v;
	
	return result;
}

// TODO: implemet INNER_EDGE to get it from the input
#define INNER_EDGE 0.046 // AU ~ 10 R_sol
__device__
var_t	gas_density_at(const planets::gaspar_t* gaspar, var_t r, var_t z)
{
	var_t result = 0.0;

	var_t h		= gaspar->sch.x * pow(r, gaspar->sch.y);
	var_t arg	= SQR(z/h);
	if (r > INNER_EDGE) {
		result	= gaspar->rho.x * pow(r, gaspar->rho.y) * exp(-arg);
	}
	else {
		var_t a	= gaspar->rho.x * pow(INNER_EDGE, gaspar->rho.y - 4.0);
		result	= a * SQR(SQR(r)) * exp(-arg);
	}

	return result;
}
#undef INNER_EDGE

inline __device__
var_t	calculate_kinetic_energy(const vec_t* velo)
{
	return 0.5 * norm2(velo);
}

inline __device__
var_t	calculate_potential_energy(var_t mu, const vec_t* coor)
{
	return -mu / norm(coor);
}

__device__
var_t	calculate_energy(const var_t mu, const vec_t* coor, const vec_t* velo)
{
	return calculate_kinetic_energy(velo) + calculate_potential_energy(mu, coor);
}

#define	sq3	1.0e-14
__device__
int		calculate_orbelem(const var_t mu, const vec_t* coor, const vec_t* velo, var_t* sma, var_t* ecc)
{
	// Calculate energy, h
    var_t h = calculate_energy(mu, coor, velo);
    if (h >= 0.0) {
        return 1;
    }

    vec_t c = cross_product(coor, velo);
	var_t cNorm2 = norm2(&c);
    /*
    * Calculate eccentricity, e
    */
    var_t e2 = 1.0 + 2.0 * cNorm2 * h / SQR(mu);
	*ecc = abs(e2) < sq3 ? 0.0 : sqrt(e2); 
    /*
    * Calculate semi-major axis, a
    */
    *sma = -mu / (2.0 * h);

    return 0;
}
#undef	sq3

#if FALSE
#define	sq2 1.0e-14
#define	sq3	1.0e-14
__device__
	int		calculate_orbelem(const var_t mu, const vec_t* coor, const vec_t* velo, planets::orbelem_t* orbelem)
{
	// Calculate energy, h
    var_t h = calculate_energy(mu, coor, velo);
    if (h >= 0.0) {
        return 1;
    }

    vec_t c = cross_product(coor, velo);
	var_t cNorm2 = norm2(&c);
    /*
    * Calculate eccentricity, e
    */
    var_t e2 = 1.0 + 2.0*cNorm2*h/SQR(mu);
	orbelem->ecc = abs(e2) < sq3 ? 0.0 : sqrt(e2); 
    /*
    * Calculate semi-major axis, a
    */
    orbelem->sma = -mu / (2.0 * h);

    /*
    * Calculate inclination, incl
    */
	cNorm2 = 2(&c);
    var_t cosi = c.z / cNorm;
    var_t sini = sqrt(c.x * c.x + c.y * c.y) / c.Length();
    var_t incl = acos(cosi);
    if (incl < sq2)
    {
        incl = 0.0;
    }
    /*
    * Calculate longitude of node, O
    */
    double node = 0.0;
    if (incl != 0.0)
    {
        double tmpx = -c.y / (c.Length() * sini);
        double tmpy = c.x / (c.Length() * sini);
		node = atan2(tmpy, tmpx);
		ShiftIntoRange(0.0, 2.0*Constants::Pi, node);
    }
    /*
    * Calculate argument of pericenter, w
    */
    double E = 0.0;
    double peri = 0.0;
    if (e2 != 0.0)
    {
        double tmpx = (l.x * cos(node) + l.y * sin(node)) / l.Length();
        double tmpy = (-l.x * sin(node) + l.y * cos(node)) / (l.Length() * cosi);
        peri = atan2(tmpy, tmpx);
        ShiftIntoRange(0.0, 2.0*Constants::Pi, peri);

        tmpx = 1.0 / e * (1.0 - r.Length() / a);
        tmpy = rv / (sqrt(mu * a) * e);
        E = atan2(tmpy, tmpx);
        ShiftIntoRange(0.0, 2.0*Constants::Pi, E);
    }
    else
    {
        peri = 0.0;
        E = atan2(r.y, r.x);
        ShiftIntoRange(0, 2.0*Constants::Pi, E);
    }
    /*
    * Calculate mean anomaly, M
    */
    double M = E - e * sin(E);
    ShiftIntoRange(0, 2.0*Constants::Pi, M);

	orbitalElement->semiMajorAxis			= a;
	orbitalElement->eccentricity			= e;
	orbitalElement->inclination				= incl;
	orbitalElement->argumentOfPericenter	= peri;
	orbitalElement->longitudeOfNode			= node;
	orbitalElement->meanAnomaly				= M;

	return 0;
}
#undef	sq2
#undef	sq3
#endif

__global__
void	calculate_grav_accel_kernel(interaction_bound iBound, const planets::param_t* params, const vec_t* coor, vec_t* acce)
{
	int	bodyIdx = iBound.sink.x + blockIdx.x * blockDim.x + threadIdx.x;

	if (bodyIdx < iBound.sink.y) {
		for (int j = iBound.source.x; j < iBound.source.y; j++) 
		{
			if (j == bodyIdx) {
				continue;
			}
			acce[bodyIdx] = calculate_grav_accel_pair(coor[bodyIdx], coor[j], params[j].mass, acce[bodyIdx]);
		}
	}
}

__global__
void calculate_drag_accel_kernel(interaction_bound iBound, var_t timeF, const planets::gaspar_t* gaspar, const planets::param_t* params, const vec_t* coor, const vec_t* velo, vec_t* acce)
{
	int	bodyIdx = iBound.sink.x + blockIdx.x * blockDim.x + threadIdx.x;

	if (bodyIdx < iBound.sink.y) {
		var_t r		= sqrt(SQR(coor[bodyIdx].x) + SQR(coor[bodyIdx].y) + SQR(coor[bodyIdx].z));
		vec_t vGas	= gas_velocity(gaspar->eta, K2*params[0].mass, r, atan2(coor[bodyIdx].y, coor[bodyIdx].x));
		var_t rhoGas= gas_density_at(gaspar, r, coor[bodyIdx].z) * timeF;

		vec_t u;
		u.x			= velo[bodyIdx].x -vGas.x;
		u.y			= velo[bodyIdx].y -vGas.y;
		u.z			= velo[bodyIdx].z -vGas.z;

		var_t C		= 0.0;
		// TODO: implement the different regimes according to the mean free path of the gas molecules
		// Epstein-regime:
		{

		}
		// Stokes-regime:
		{
			var_t uLength = sqrt(SQR(vGas.x) + SQR(vGas.y) + SQR(vGas.z));
			C = params[bodyIdx].gamma_stokes * uLength * rhoGas;
		}
		// Transition regime:
		{

		}

		acce[bodyIdx].x = -C * u.x;
		acce[bodyIdx].y = -C * u.y;
		acce[bodyIdx].z = -C * u.z;
	}
}

hipError_t	planets::call_calculate_grav_accel_kernel(number_of_bodies nBodies, const planets::param_t* params, const vec_t* coor, vec_t* acce)
{
	hipError_t cudaStatus = hipSuccess;
	
	interaction_bound iBound = nBodies.get_self_interacting();

	int		nBodyToCalculate = nBodies.n_self_interacting();
	int		nThread = std::min(THREADS_PER_BLOCK, nBodyToCalculate);
	int		nBlock = (nBodyToCalculate + nThread - 1)/nThread;
	dim3	grid(nBlock);
	dim3	block(nThread);

	calculate_grav_accel_kernel<<<grid, block>>>(iBound, params, coor, acce);
	if ((cudaStatus = hipGetLastError()) != hipSuccess) {
		throw nbody_exception("calculate_grav_accel_kernel launch failed", cudaStatus);
	}

	iBound = nBodies.get_nonself_interacting();
	nBodyToCalculate = nBodies.super_planetesimal + nBodies.planetesimal;
	nThread		= std::min(THREADS_PER_BLOCK, nBodyToCalculate);
	nBlock		= (nBodyToCalculate + nThread - 1)/nThread;
	grid.x		= nBlock;
	block.x		= nThread;

	calculate_grav_accel_kernel<<<grid, block>>>(iBound, params, coor, acce);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		throw nbody_exception("calculate_grav_accel_kernel launch failed", cudaStatus);
	}

	iBound = nBodies.get_non_interacting();
	nBodyToCalculate = nBodies.test_particle;
	nThread		= std::min(THREADS_PER_BLOCK, nBodyToCalculate);
	nBlock		= (nBodyToCalculate + nThread - 1)/nThread;
	grid.x		= nBlock;
	block.x		= nThread;

	calculate_grav_accel_kernel<<<grid, block>>>(iBound, params, coor, acce);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		throw nbody_exception("calculate_grav_accel_kernel launch failed", cudaStatus);
	}

	return cudaStatus;
}

hipError_t planets::call_calculate_drag_accel_kernel(number_of_bodies nBodies, ttt_t time, const planets::gaspar_t* gaspar, const planets::param_t* params, const vec_t* coor, const vec_t* velo, vec_t* acce)
{
	hipError_t cudaStatus = hipSuccess;

	// TODO: calculate it using the value of the time
	var_t timeF = 1.0;
	
	interaction_bound iBound = nBodies.get_bodies_gasdrag();

	int		nBodyToCalculate = nBodies.super_planetesimal + nBodies.planetesimal;
	int		nThread = std::min(THREADS_PER_BLOCK, nBodyToCalculate);
	int		nBlock = (nBodyToCalculate + nThread - 1)/nThread;
	dim3	grid(nBlock);
	dim3	block(nThread);

	calculate_drag_accel_kernel<<<grid, block>>>(iBound, timeF, gaspar, params, coor, velo, acce);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		throw nbody_exception("calculate_drag_accel_kernel launch failed", cudaStatus);
	}

	return cudaStatus;
}

planets::planets(number_of_bodies bodies) :
	ode(2),
	bodies(bodies)
{
	round_up_n();
	allocate_vectors();
}

planets::~planets()
{
}

void planets::allocate_vectors()
{
	// Parameters
	h_p.resize(NPAR * bodies.total);

	// Aliases to coordinates and velocities
	h_y[0].resize(NDIM * bodies.total);
	h_y[1].resize(NDIM * bodies.total);
}

void planets::round_up_n()
{
	// Round up n to the number of bodies per tile
	int m = ((bodies.total + NTILE - 1) / NTILE) * NTILE;

	if (bodies.total != m) {
		cerr << "Number of bodies rounded up to " << m << endl;
	}
	bodies.total_rounded_up = m;
}

