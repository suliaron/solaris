#include "hip/hip_runtime.h"
// includes, system 
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <string>

#include "Constants.h"
#include "number_of_bodies.h"

#define PI 3.1415926535897932384626
#define K2 0.0002959122082855911025

#define MASS_STAR		1.0			// M_sol
#define MASS_JUPITER	1.0e-3		// M_sol
#define RAD_STAR		0.005		// AU

#define MASS_SUN		1.9891E+30	// kg
#define MASS_FACTOR		5e-7		// M_sol
#define MASS_MU			log(4.0)
#define MASS_S			0.3
#define MASS_MIN		1.0e-20		// M_sol
#define MASS_MAX		1.0e-19		// M_sol

#define DIST_MIN		4.5			// AU
#define DIST_MAX		15			// AU

#define DENSITY			3000.0		// kg m-3
#define AU				149.6e9		// m

// It must be enclosed in parentheses in order to give correct results in
// the case of a division i.e. 1/SQR(x) -> 1/((x)*(x))
#define	SQR(x)			((x)*(x))
#define	CUBE(x)			((x)*(x)*(x))

using namespace std;

typedef double	var_t;
typedef int		int_t;

typedef struct double2
{
	var_t	x;
	var_t	y;
} var2_t;

typedef struct double4
{
	var_t	x;
	var_t	y;
	var_t	z;
	var_t	w;
} vec_t;

typedef struct orbelem
{
	//! Semimajor-axis of the body
	var_t sma;
	//! Eccentricity of the body
	var_t ecc;
	//! Inclination of the body
	var_t inc;
	//! Argument of the pericenter
	var_t peri;
	//! Longitude of the ascending node
	var_t node;
	//! Mean anomaly
	var_t mean;
} orbelem_t;

typedef struct param
{
	//! Mass of body in M_sol
	var_t mass;
	//! Radius of body in AU
	var_t radius;
	//! Density of body in M_sol AU-3
	var_t density;
	//! Drag coefficient, used to compute the stokes drag force
	var_t cd;
} param_t;

// Draw a number from a given distribution
var_t generate_random(var_t xmin, var_t xmax, var_t p(var_t))
{
	var_t x;
	var_t y;

	do
	{
		x = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
		y = (var_t)rand() / RAND_MAX;
	}
	while (y > p(x));

	return x;
}

var_t pdf_mass_lognormal(var_t x)
{
	return 1.0 / sqrt(2 * PI) / MASS_S * exp(-pow(log(x) - MASS_MU, 2) / 2 / MASS_S / MASS_MU);
}

var_t pdf_distance_squared(var_t d)
{
	return d * d / DIST_MAX / DIST_MAX;
}

var_t pdf_distance_exp(var_t d)
{
	return exp(-d) * d * d;
}

var_t pdf_const(var_t x)
{
	return 1;
}

void calculate_circle_coord(var_t d, var_t phi, var_t* x, var_t* y)
{
	*x = d * cos(phi);
	*y = d * sin(phi);
}

void calculate_circle_veloc(var_t d, var_t phi, var_t* vx, var_t* vy)
{
	var_t v = sqrt(K2 * MASS_STAR / d);
	
	*vx = v * sin(phi);
	*vy = - v * cos(phi);
}

var_t calculate_radius(var_t m)
{
	var_t V = m * MASS_SUN / DENSITY;	// m3
	V /= AU * AU * AU;		// AU3
	
	return pow(3.0 / 4.0 / PI * V, 1.0 / 3.0);
}

#define FOUR_PI_OVER_THREE	4.1887902047863909846168578443727
var_t calculate_radius(var_t m, var_t density)
{
	return pow(1.0/FOUR_PI_OVER_THREE * m/density ,1.0/3.0);
}

var_t calculate_density(var_t m, var_t R)
{
	return m / (FOUR_PI_OVER_THREE * CUBE(R));
}

var_t caclulate_mass(var_t R, var_t density)
{
	return FOUR_PI_OVER_THREE * CUBE(R) * density;
}
#undef FOUR_PI_OVER_THREE

int_t	kepler_equation_solver(var_t ecc, var_t mean, var_t eps, var_t* E)
{
	if (ecc == 0.0 || mean == 0.0 || mean == PI) {
        *E = mean;
		return 0;
    }
    *E = mean + ecc * (sin(mean)) / (1.0 - sin(mean + ecc) + sin(mean));
    var_t E1 = 0.0;
    var_t error;
    int_t step = 0;
    do {
        E1 = *E - (*E - ecc * sin(*E) - mean) / (1.0 - ecc * cos(*E));
        error = fabs(E1 - *E);
        *E = E1;
    } while (error > eps && step++ <= 15);
	if (step > 15 ) {
		return 1;
	}

	return 0;
}

int_t	calculate_phase(var_t mu, const orbelem_t* oe, vec_t* rVec, vec_t* vVec)
{
    var_t ecc = oe->ecc;
	var_t E = 0.0;
	if (kepler_equation_solver(ecc, oe->mean, 1.0e-14, &E) == 1) {
		return 1;
	}
    var_t v = 2.0 * atan(sqrt((1.0 + ecc) / (1.0 - ecc)) * tan(E / 2.0));

    var_t p = oe->sma * (1.0 - SQR(ecc));
    var_t r = p / (1.0 + ecc * cos(v));
    var_t kszi = r * cos(v);
    var_t eta = r * sin(v);
    var_t vKszi = -sqrt(mu / p) * sin(v);
    var_t vEta = sqrt(mu / p) * (ecc + cos(v));

    var_t cw = cos(oe->peri);
    var_t sw = sin(oe->peri);
    var_t cO = cos(oe->node);
    var_t sO = sin(oe->node);
    var_t ci = cos(oe->inc);
    var_t si = sin(oe->inc);

    vec_t P;
	P.x = cw * cO - sw * sO * ci;
	P.y = cw * sO + sw * cO * ci;
	P.z = sw * si;
    vec_t Q;
	Q.x = -sw * cO - cw * sO * ci;
	Q.y = -sw * sO + cw * cO * ci;
	Q.z = cw * si;

	rVec->x = kszi * P.x + eta * Q.x;
	rVec->y = kszi * P.y + eta * Q.y;
	rVec->z = kszi * P.z + eta * Q.z;

	vVec->x = vKszi * P.x + vEta * Q.x;
	vVec->y = vKszi * P.y + vEta * Q.y;
	vVec->z = vKszi * P.z + vEta * Q.z;

	return 0;
}

string combine_path(string dir, string filename)
{
	if (dir.size() > 0) {
		if (*(dir.end() - 1) != '/' && *(dir.end() - 1) != '\\') {
			return dir + '/' + filename;
		}
		else {
			return dir + filename;
		}
	}
	else {
		return filename;
	}
}

int generate_nbody(string filename, int n)
{
	var_t d;
	var_t phi;
	var_t m;

	var_t x, y, z;
	var_t vx, vy, vz;
	var_t r;

	char sep = ' ';

	std::ofstream	output;
	output.open(filename, std::ios_base::app);

	// Output central mass
	output << 0 << sep;
	output << 0.0 << sep;
	output << 1.0 << sep << Constants::SolarRadiusToAu << sep;
	output << 0.0 << sep << 0.0 << sep << 0.0 << sep;
	output << 0.0 << sep << 0.0 << sep << 0.0;
	output << endl;

	// Output planets
	for (int i = 1; i < n; i ++)
	{
		d = generate_random(DIST_MIN, DIST_MAX, pdf_distance_squared);
		phi = generate_random(0, 2*PI, pdf_const);
		m = 1.0e-19; //MASS_FACTOR * generate_random(MASS_MIN, MASS_MAX, pdf_mass_lognormal);

		calculate_circle_coord(d, phi, &x, &y);
		calculate_circle_veloc(d, phi, &vx, &vy);
		r = calculate_radius(m);
		z = vz = 0;

		output << i << sep;
		output << 0 << sep;
		output << m << sep << r << sep;
		output << x << sep << y << sep << z << sep;
		output << vx << sep << vy << sep << vz;
		output << endl;
	}

	return 0;
}

int generate_nbody_Rezso(string filename, int n)
{
	var_t m, m0, m1;
	var_t r;

	char sep = ' ';

	std::ofstream	output;
	output.open(filename, std::ios_base::app);

	// Output central mass
	m0 = 2.0;
	output << 0 << sep;
	output << 0.0 << sep;
	output << m0 << sep << 1.2 * Constants::SolarRadiusToAu << sep;
	output << 0.0 << sep << 0.0 << sep << 0.0 << sep;
	output << 0.0 << sep << 0.0 << sep << 0.0;
	output << endl;

	// Calculate phase of giant planet
	orbelem_t oe = {64.5, 0.0, 0.0, 0.0, 0.0, 0.0};
	vec_t rVec = {0.0, 0.0, 0.0, 0.0};
	vec_t vVec = {0.0, 0.0, 0.0, 0.0};
	m1 = 5.0 * Constants::JupiterToSolar;
	var_t mu = K2*(m0 + m1);
	int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
	if (ret_code == 1) {
		cerr << "Could not calculate the phase." << endl;
		return ret_code;
	}

	// Output giant planet
	output << 1 << sep;
	output << 0.0 << sep;
	output << m1 << sep << 1.0e-1 * Constants::SolarRadiusToAu << sep;
	output << rVec.x << sep << rVec.y << sep << rVec.z << sep;
	output << vVec.x << sep << vVec.y << sep << vVec.z << sep;
	output << endl;

	srand (time(0));
	// Output planets
	for (int i = 2; i < n; i ++)
	{
		oe.sma = generate_random(70.0, 270.0, pdf_const);
		oe.ecc = 0.0;
		oe.inc = atan(0.05); // tan(i) = h/r = 5.0e-2
		oe.peri = generate_random(0.0, 2.0*PI, pdf_const);
		oe.node = generate_random(0.0, 2.0*PI, pdf_const);
		oe.mean = generate_random(0.0, 2.0*PI, pdf_const);

		var_t mu = K2*(m0 + 0.0);
		int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
		if (ret_code == 1) {
			cerr << "Could not calculate the phase." << endl;
			return ret_code;
		}

		m = 0.0;
		r = 0.0;

		output << i << sep;
		output << 0 << sep;
		output << m << sep << r << sep;
		output << rVec.x << sep << rVec.y << sep << rVec.z << sep;
		output << vVec.x << sep << vVec.y << sep << vVec.z << sep;
		output << endl;
	}

	return 0;
}

int generate_nbody2(string filename, int n)
{
	var_t m, m0, m1;
	var_t r;

	char sep = ' ';

	std::ofstream	output;
	output.open(filename, std::ios_base::app);

	// Output central mass
	m0 = 1.0;
	output << 0 << sep;
	output << 0.0 << sep;
	output << m0 << sep << Constants::SolarRadiusToAu << sep;
	output << 0.0 << sep << 0.0 << sep << 0.0 << sep;
	output << 0.0 << sep << 0.0 << sep << 0.0;
	output << endl;

	srand (time(0));
	orbelem oe;
	vec_t	rVec, vVec;
	// Output planets
	for (int i = 1; i < n; i ++)
	{
		oe.sma = generate_random(0.5, 10.0, pdf_const);
		oe.ecc = 0.0;
		oe.inc = atan(0.05); // tan(i) = h/r = 5.0e-2
		oe.peri = generate_random(0.0, 2.0*PI, pdf_const);
		oe.node = generate_random(0.0, 2.0*PI, pdf_const);
		oe.mean = generate_random(0.0, 2.0*PI, pdf_const);

		var_t mu = K2*(m0 + 0.0);
		int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
		if (ret_code == 1) {
			cerr << "Could not calculate the phase." << endl;
			return ret_code;
		}

		m = 1.0e-6;
		r = 1.0e-6;

		output << i << sep;
		output << 0 << sep;
		output << m << sep << r << sep;
		output << rVec.x << sep << rVec.y << sep << rVec.z << sep;
		output << vVec.x << sep << vVec.y << sep << vVec.z << sep;
		output << endl;
	}

	return 0;
}

void print_body_record(std::ofstream &output, int_t bodyId, var_t t, param_t *param, vec_t *r, vec_t *v)
{
	static char sep = ' ';

	output << bodyId << sep << t << sep;
	output << param->mass << sep << param->radius << sep << param->density << sep << param->cd << sep;
	output << r->x << sep << r->y << sep << r->z << sep;
	output << v->x << sep << v->y << sep << v->z << sep;
	output << endl;
}

int generate_pp_disk(string filename, var2_t disk, number_of_bodies *nBodies)
{
	var_t t = 0.0;
	int_t bodyId = 0;

	param_t param0;
	param_t param;
	vec_t	rVec = {0.0, 0.0, 0.0, 0.0};
	vec_t	vVec = {0.0, 0.0, 0.0, 0.0};

	std::ofstream	output;
	output.open(filename, std::ios_base::app);

	// Output central mass
	for (int i = 0; i < nBodies->star; i++, bodyId++)
	{
		param0.mass = 1.0;
		param0.radius = Constants::SolarRadiusToAu;
		param0.density = calculate_density(param0.mass, param0.radius);
		param0.cd = 0.0;
		print_body_record(output, bodyId, t, &param0, &rVec, &vVec);
	}

	srand (time(0));
	orbelem oe;
	// Output giant planets
	for (int i = 0; i < nBodies->giant_planet; i++, bodyId++)
	{
		oe.sma = generate_random(disk.x, disk.y, pdf_const);
		oe.ecc = generate_random(0.0, 0.1, pdf_const);
		oe.inc = atan(0.05); // tan(i) = h/r = 5.0e-2
		oe.peri = generate_random(0.0, 2.0*PI, pdf_const);
		oe.node = generate_random(0.0, 2.0*PI, pdf_const);
		oe.mean = generate_random(0.0, 2.0*PI, pdf_const);

		param.mass = generate_random(0.1, 10.0, pdf_const) * Constants::JupiterToSolar;
		param.density = generate_random(1.0, 2.0, pdf_const) * Constants::GramPerCm3ToSolarPerAu3;
		param.radius = calculate_radius(param.mass, param.density);
		param.cd = 0.0;

		var_t mu = K2*(param0.mass + param.mass);
		int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
		if (ret_code == 1) {
			cerr << "Could not calculate the phase." << endl;
			return ret_code;
		}
		print_body_record(output, bodyId, t, &param, &rVec, &vVec);
	}

	// Output rocky planets
	for (int i = 0; i < nBodies->rocky_planet; i++, bodyId++)
	{
		oe.sma = generate_random(disk.x, disk.y, pdf_const);
		oe.ecc = generate_random(0.0, 0.1, pdf_const);
		oe.inc = atan(0.05); // tan(i) = h/r = 5.0e-2
		oe.peri = generate_random(0.0, 2.0*PI, pdf_const);
		oe.node = generate_random(0.0, 2.0*PI, pdf_const);
		oe.mean = generate_random(0.0, 2.0*PI, pdf_const);

		param.mass = generate_random(0.1, 10.0, pdf_const) * Constants::EarthToSolar;
		param.density = generate_random(3.0, 5.5, pdf_const) * Constants::GramPerCm3ToSolarPerAu3;
		param.radius = calculate_radius(param.mass, param.density);
		param.cd = 0.0;

		var_t mu = K2*(param0.mass + param.mass);
		int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
		if (ret_code == 1) {
			cerr << "Could not calculate the phase." << endl;
			return ret_code;
		}
		print_body_record(output, bodyId, t, &param, &rVec, &vVec);
	}

	// Output proto planets
	for (int i = 0; i < nBodies->proto_planet; i++, bodyId++)
	{
		oe.sma = generate_random(disk.x, disk.y, pdf_const);
		oe.ecc = generate_random(0.0, 0.1, pdf_const);
		oe.inc = atan(0.05); // tan(i) = h/r = 5.0e-2
		oe.peri = generate_random(0.0, 2.0*PI, pdf_const);
		oe.node = generate_random(0.0, 2.0*PI, pdf_const);
		oe.mean = generate_random(0.0, 2.0*PI, pdf_const);

		param.mass = generate_random(0.001, 0.1, pdf_const) * Constants::EarthToSolar;
		param.density = generate_random(1.5, 3.5, pdf_const) * Constants::GramPerCm3ToSolarPerAu3;
		param.radius = calculate_radius(param.mass, param.density);
		param.cd = 0.0;

		var_t mu = K2*(param0.mass + param.mass);
		int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
		if (ret_code == 1) {
			cerr << "Could not calculate the phase." << endl;
			return ret_code;
		}
		print_body_record(output, bodyId, t, &param, &rVec, &vVec);
	}

	// Output super-planetesimals
	for (int i = 0; i < nBodies->super_planetesimal; i++, bodyId++)
	{
		oe.sma = generate_random(disk.x, disk.y, pdf_const);
		oe.ecc = generate_random(0.0, 0.2, pdf_const);
		oe.inc = atan(0.05); // tan(i) = h/r = 5.0e-2
		oe.peri = generate_random(0.0, 2.0*PI, pdf_const);
		oe.node = generate_random(0.0, 2.0*PI, pdf_const);
		oe.mean = generate_random(0.0, 2.0*PI, pdf_const);

		param.mass = generate_random(0.0001, 0.01, pdf_const) * Constants::EarthToSolar;
		param.density = generate_random(1.0, 2.0, pdf_const) * Constants::GramPerCm3ToSolarPerAu3;
		param.radius = generate_random(5.0, 15.0) * Constants::KilometerToAu;
		param.cd = generate_random(0.5, 4.0);

		var_t mu = K2*(param0.mass + param.mass);
		int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
		if (ret_code == 1) {
			cerr << "Could not calculate the phase." << endl;
			return ret_code;
		}
		print_body_record(output, bodyId, t, &param, &rVec, &vVec);
	}

	// Output planetesimals
	for (int i = 0; i < nBodies->planetesimal; i++, bodyId++)
	{
		oe.sma = generate_random(disk.x, disk.y, pdf_const);
		oe.ecc = generate_random(0.0, 0.2, pdf_const);
		oe.inc = atan(0.05); // tan(i) = h/r = 5.0e-2
		oe.peri = generate_random(0.0, 2.0*PI, pdf_const);
		oe.node = generate_random(0.0, 2.0*PI, pdf_const);
		oe.mean = generate_random(0.0, 2.0*PI, pdf_const);

		param.density = generate_random(1.0, 2.0, pdf_const) * Constants::GramPerCm3ToSolarPerAu3;
		param.radius = generate_random(5.0, 15.0) * Constants::KilometerToAu;
		param.mass = caclulate_mass(param.radius, param.density);
		param.cd = generate_random(0.5, 4.0);

		var_t mu = K2*(param0.mass + param.mass);
		int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
		if (ret_code == 1) {
			cerr << "Could not calculate the phase." << endl;
			return ret_code;
		}
		print_body_record(output, bodyId, t, &param, &rVec, &vVec);
	}

	// Output test particles
	for (int i = 0; i < nBodies->test_particle; i++, bodyId++)
	{
		oe.sma = generate_random(disk.x, disk.y, pdf_const);
		oe.ecc = generate_random(0.0, 0.2, pdf_const);
		oe.inc = atan(0.05); // tan(i) = h/r = 5.0e-2
		oe.peri = generate_random(0.0, 2.0*PI, pdf_const);
		oe.node = generate_random(0.0, 2.0*PI, pdf_const);
		oe.mean = generate_random(0.0, 2.0*PI, pdf_const);

		param.density = 0.0;
		param.radius = 0.0;
		param.mass = 0.0;
		param.cd = 0.0;

		var_t mu = K2*(param0.mass + param.mass);
		int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
		if (ret_code == 1) {
			cerr << "Could not calculate the phase." << endl;
			return ret_code;
		}
		print_body_record(output, bodyId, t, &param, &rVec, &vVec);
	}

	return 0;
}

int generate_2_body(string filename, int n)
{
	var_t m, m0, m1;
	var_t r;

	char sep = ' ';

	std::ofstream	output;
	output.open(filename, ios::trunc);

	// Output central mass
	m0 = 1.0;
	output << 0 << sep;
	output << 0.0 << sep;
	output << m0 << sep << Constants::SolarRadiusToAu << sep;
	output << 0.0 << sep << 0.0 << sep << 0.0 << sep;
	output << 0.0 << sep << 0.0 << sep << 0.0 << endl;

	// Calculate phase of super-planetesimal
	var_t s = Constants::DegreeToRadian;
	orbelem_t oe = {5.20336301, 0.04839266, 1.3053 * s, 274.1977 * s, 100.55615 * s, 19.65053 * s};
	vec_t rVec = {0.0, 0.0, 0.0, 0.0};
	vec_t vVec = {0.0, 0.0, 0.0, 0.0};
	m1 = Constants::JupiterToSolar;
	var_t mu = Constants::Gauss2*(m0 + m1);
	int_t ret_code = calculate_phase(mu, &oe, &rVec, &vVec);
	if (ret_code == 1) {
		cerr << "Could not calculate the phase." << endl;
		return ret_code;
	}

	// Output super-planetesimal
	output << 1 << sep;
	output << 0.0 << sep;
	output << m1 << sep << 1.0e-5 * Constants::SolarRadiusToAu << sep;
	output << setprecision(16) << rVec.x << sep << setprecision(16) << rVec.y << sep << setprecision(16) << rVec.z << sep;
	output << setprecision(16) << vVec.x << sep << setprecision(16) << vVec.y << sep << setprecision(16) << vVec.z << sep;
	output << endl;

	return 0;
}

int parse_options(int argc, const char** argv, number_of_bodies *nBodies, int *n, string &nBodies_str)
{
	int i = 1;

	while (i < argc) {
		string p = argv[i];

		// Number of bodies
		if (p == "-n") {
			i++;
			*n = atoi(argv[i]);
			if (2 > *n) {
				cerr << "Number of bodies must exceed 2.";
				return 1;
			}
		}
		else if (p == "-nBodies") {
			i++;
			int iSav = i;
			int	star				= atoi(argv[i++]);
			int	giant_planet		= atoi(argv[i++]);
			int	rocky_planet		= atoi(argv[i++]);
			int	proto_planet		= atoi(argv[i++]);
			int	super_planetesimal	= atoi(argv[i++]);
			int	planetesimal		= atoi(argv[i++]);
			int	test_particle		= atoi(argv[i]);
			nBodies = new number_of_bodies(star, giant_planet, rocky_planet, proto_planet, super_planetesimal, planetesimal, test_particle);

			string number;
			for (int k = iSav; k < iSav + 6; iSav++) {
				number = argv[k];
				nBodies_str += number + '_';
			}
			number = argv[i+6];
			nBodies_str += number;
			break;
		}
		else {
			cerr << "Invalid switch on command-line.";
			return 1;
		}
		i++;
	}

	return 0;
}

int main(int argc, char* argv[])
{
	int n = 0;
	number_of_bodies *nBodies = 0;
	string nBodies_str;

	int retCode = parse_options(argc, argv, nBodies, &n, nBodies_str);
	if (0 != retCode) {
		exit(retCode);
	}
	

	const string baseDir = "C:\\Work\\Solaris.Cuda.TestRuns";
	const string subDir = "256_Body";
	string curDir = combine_path(baseDir, subDir);
	
	srand(time(NULL));

	//retCode = generate_nbody2(combine_path(curDir, "256_Body.txt"), 256);
	//retCode = generate_nbody_Rezso("E:\\Work\\VSSolutions\\solaris\\src\\Solaris.NBody.Cuda.Test\\TestRun\\Rezso\\Rezso.txt", n);
	//retCode = generate_2_body(combine_path(curDir, "TwoBody.txt"), 2);

	retCode = generate_pp_disk(combine_path(curDir, ("nBodies_" + nBodies_str + ".txt")), nBodies);

	return retCode;
}
